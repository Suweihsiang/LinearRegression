#include "hip/hip_runtime.h"
#include"utils.cuh"

__global__ void multi_matrix(double* as, double* bs, double* cs, int row, int col, int k) {
	int r = threadIdx.y + blockDim.y * blockIdx.y;
	int c = threadIdx.x + blockDim.x * blockIdx.x;
	if (r < row && c < col) {
		double val = 0.0;
		for (int i = 0; i < k; i++) {
			val += as[r + i * row] * bs[c * k + i];
		}
		cs[r + c * row] = val;
	}
}

hipError_t matmul(MatrixXd& a, MatrixXd& b, MatrixXd& c) {
	double* as;
	double* bs;
	double* cs;
	size_t a_sz = sizeof(double) * a.size();
	size_t b_sz = sizeof(double) * b.size();
	size_t c_sz = sizeof(double) * c.size();
	hipError_t hipError_t;
	hipError_t = hipMalloc(&as, a_sz);
	hipError_t = hipMalloc(&bs, b_sz);
	hipError_t = hipMalloc(&cs, c_sz);
	hipError_t = hipMemcpy(as, a.data(), a_sz, hipMemcpyHostToDevice);
	hipError_t = hipMemcpy(bs, b.data(), b_sz, hipMemcpyHostToDevice);
	dim3 threads_per_block(32, 32, 1);
	dim3 blocks_per_grid((b.cols() + 32 - 1) / 32, (a.rows() + 32 - 1) / 32, 1);
	multi_matrix << <blocks_per_grid, threads_per_block >> > (as, bs, cs, a.rows(), b.cols(), a.cols());
	hipError_t = hipGetLastError();
	hipError_t = hipDeviceSynchronize();
	hipError_t = hipMemcpy(c.data(), cs, c_sz, hipMemcpyDeviceToHost);
	hipError_t = hipFree(as);
	hipError_t = hipFree(bs);
	hipError_t = hipFree(cs);
	return hipError_t;
}

__global__ void multi_matrix_shared(double* a,double* b, double* c, int row, int col, int k) {
	const int BM = 128;
	const int BN = 128;
	const int BK = 8;
	const int TM = 8;
	const int TN = 8;
	
	__shared__ double a_shared[BM][BK];
	__shared__ double b_shared[BK][BN];
	double sub_c[TM][TN] = { 0.0 };

	const int by = blockIdx.y;
	const int bx = blockIdx.x;
	const int ty = threadIdx.y;
	const int tx = threadIdx.x;
	const int tid = ty * blockDim.x + tx;

	int a_shared_row = tid >> 1;
	int a_shared_col = (tid % 2) << 2;

	int b_shared_row = tid >> 5;
	int b_shared_col = (tid % 32) << 2;

	int a_global_row = by * BM + a_shared_row;
	int b_global_col = bx * BN + b_shared_col;

	for (int bk = 0; bk < (k + BK - 1) / BK; bk++) {
		int a_global_col = bk * BK + a_shared_col;
		int a_global_site = a_global_row + a_global_col * row;

		int b_global_row = bk * BK + b_shared_row;
		int b_global_site = b_global_row + b_global_col * k;
		for (int i = 0; i < 4; i++) {
			a_shared[a_shared_row][a_shared_col + i] = (a_global_row < row && a_global_col < k) ? a[a_global_site + i * row] : 0;
			b_shared[b_shared_row][b_shared_col + i] = (b_global_row < k && b_global_col < col) ? b[b_global_site + i * k] : 0;
		}
		__syncthreads();
		
		#pragma unroll
		for (int j = 0; j < BK; j++) {
			#pragma unroll
			for (int m = 0; m < TM; m++) {
				int a_shared_inc_row = ty * TM + m;
				#pragma unroll
				for (int n = 0; n < TN; n++) {
					int b_shared_inc_col = tx * TN + n;
					sub_c[m][n] += a_shared[a_shared_inc_row][j] * b_shared[j][b_shared_inc_col];
				}
			}
		}
		__syncthreads();
	}
	#pragma unroll
	for (int m = 0; m < TM; m++) {
		int c_global_row = by * BM + ty * TM + m;
		if (c_global_row >= row) { break; }
		#pragma unroll
		for (int n = 0; n < TN; n += 4) {
			int c_global_col = bx * BN + tx * TN + n;
			if (c_global_col >= col) { break; }
			int c_global_site = c_global_row + c_global_col * row;
			for (int i = 0; i < 4; i++) {
				if (c_global_col + i >= col) { break; }
				c[c_global_site + i * row] = sub_c[m][n + i];
			}
		}
	}
}

hipError_t matmul_shared(MatrixXd& a, MatrixXd& b, MatrixXd& c) {
	double* aptr;
	double* bptr;
	double* cptr;
	size_t a_size = sizeof(double) * a.size();
	size_t b_size = sizeof(double) * b.size();
	size_t c_size = sizeof(double) * c.size();
	hipError_t hipError_t;
	hipError_t = hipMalloc(&aptr, a_size);
	hipError_t = hipMalloc(&bptr, b_size);
	hipError_t = hipMalloc(&cptr, c_size);
	hipError_t = hipMemcpy(aptr, a.data(), a_size, hipMemcpyHostToDevice);
	hipError_t = hipMemcpy(bptr, b.data(), b_size, hipMemcpyHostToDevice);
	dim3 blocksdim(16, 16, 1);
	dim3 gridsdim((b.cols() - 1) / 128 + 1, (a.rows() - 1) / 128 + 1, 1);
	multi_matrix_shared << <gridsdim, blocksdim >> > (aptr, bptr, cptr, a.rows(), b.cols(), a.cols());
	hipError_t = hipGetLastError();
	hipError_t = hipDeviceSynchronize();
	hipError_t = hipMemcpy(c.data(), cptr, c_size, hipMemcpyDeviceToHost);
	hipError_t = hipFree(aptr);
	hipError_t = hipFree(bptr);
	hipError_t = hipFree(cptr);
	return hipError_t;
}

__global__ void add_matrix(double* a, double* b, double* c, int row, int col) {
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	if (x < col && y < row) {
		c[y + x * row] = a[y + x * row] + b[y + x * row];
	}
}

hipError_t matadd(MatrixXd& a, MatrixXd& b, MatrixXd& c) {
	double* aptr;
	double* bptr;
	double* cptr;
	size_t a_size = sizeof(double) * a.size();
	hipError_t hipError_t;
	hipError_t = hipMalloc(&aptr, a_size);
	hipError_t = hipMalloc(&bptr, a_size);
	hipError_t = hipMalloc(&cptr, a_size);
	hipError_t = hipMemcpy(aptr, a.data(), a_size, hipMemcpyHostToDevice);
	hipError_t = hipMemcpy(bptr, b.data(), a_size, hipMemcpyHostToDevice);
	dim3 blockdim(32, 32, 1);
	dim3 griddim((a.cols() - 1) / 32 + 1, (a.rows() - 1) / 32 + 1, 1);
	add_matrix << <griddim, blockdim >> > (aptr, bptr, cptr, a.rows(), a.cols());
	hipError_t = hipGetLastError();
	hipError_t = hipDeviceSynchronize();
	hipError_t = hipMemcpy(c.data(), cptr, a_size, hipMemcpyDeviceToHost);
	hipError_t = hipFree(aptr);
	hipError_t = hipFree(bptr);
	hipError_t = hipFree(cptr);
	return hipError_t;
}