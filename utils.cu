#include "hip/hip_runtime.h"
#include"utils.cuh"
#include<stdio.h>
#include<stdlib.h>

#define CHECK_CUDA_ERROR(val) Check_cuda_Error((val),__FILE__,__LINE__)
void Check_cuda_Error(hipError_t error, const char* const file, const int line) {
	if (error != hipSuccess) {
		printf("CUDA Error Occurs at %s, line %d : #%d(%s)\n", file, line, error, hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}
}

__global__ void multi_matrix(double* as, double* bs, double* cs, int row, int col, int k) {
	int r = threadIdx.y + blockDim.y * blockIdx.y;
	int c = threadIdx.x + blockDim.x * blockIdx.x;
	if (r < row && c < col) {
		double val = 0.0;
		for (int i = 0; i < k; i++) {
			val += as[r + i * row] * bs[c * k + i];
		}
		cs[r + c * row] = val;
	}
}

void matmul(MatrixXd& a, MatrixXd& b, MatrixXd& c) {
	double* as;
	double* bs;
	double* cs;
	size_t a_sz = sizeof(double) * a.size();
	size_t b_sz = sizeof(double) * b.size();
	size_t c_sz = sizeof(double) * c.size();
	CHECK_CUDA_ERROR(hipMalloc(&as, a_sz));
	CHECK_CUDA_ERROR(hipMalloc(&bs, b_sz));
	CHECK_CUDA_ERROR(hipMalloc(&cs, c_sz));
	CHECK_CUDA_ERROR(hipMemcpy(as, a.data(), a_sz, hipMemcpyHostToDevice));
	CHECK_CUDA_ERROR(hipMemcpy(bs, b.data(), b_sz, hipMemcpyHostToDevice));
	dim3 threads_per_block(32, 32, 1);
	dim3 blocks_per_grid((b.cols() + 32 - 1) / 32, (a.rows() + 32 - 1) / 32, 1);
	multi_matrix << <blocks_per_grid, threads_per_block >> > (as, bs, cs, a.rows(), b.cols(), a.cols());
	CHECK_CUDA_ERROR(hipDeviceSynchronize());
	CHECK_CUDA_ERROR(hipMemcpy(c.data(), cs, c_sz, hipMemcpyDeviceToHost));
	CHECK_CUDA_ERROR(hipFree(as));
	CHECK_CUDA_ERROR(hipFree(bs));
	CHECK_CUDA_ERROR(hipFree(cs));
	return ;
}

__global__ void multi_matrix_shared(double* a,double* b, double* c, int row, int col, int k) {
	const int BM = 128;
	const int BN = 128;
	const int BK = 8;
	const int TM = 8;
	const int TN = 8;
	
	__shared__ double a_shared[BM][BK];
	__shared__ double b_shared[BK][BN];
	double sub_c[TM][TN] = { 0.0 };

	const int by = blockIdx.y;
	const int bx = blockIdx.x;
	const int ty = threadIdx.y;
	const int tx = threadIdx.x;
	const int tid = ty * blockDim.x + tx;

	int a_shared_row = tid >> 1;
	int a_shared_col = (tid % 2) << 2;

	int b_shared_row = tid >> 5;
	int b_shared_col = (tid % 32) << 2;

	int a_global_row = by * BM + a_shared_row;
	int b_global_col = bx * BN + b_shared_col;

	for (int bk = 0; bk < (k + BK - 1) / BK; bk++) {
		int a_global_col = bk * BK + a_shared_col;
		int a_global_site = a_global_row + a_global_col * row;

		int b_global_row = bk * BK + b_shared_row;
		int b_global_site = b_global_row + b_global_col * k;
		for (int i = 0; i < 4; i++) {
			a_shared[a_shared_row][a_shared_col + i] = (a_global_row < row && a_global_col < k) ? a[a_global_site + i * row] : 0;
			b_shared[b_shared_row][b_shared_col + i] = (b_global_row < k && b_global_col < col) ? b[b_global_site + i * k] : 0;
		}
		__syncthreads();
		
		#pragma unroll
		for (int j = 0; j < BK; j++) {
			#pragma unroll
			for (int m = 0; m < TM; m++) {
				int a_shared_inc_row = ty * TM + m;
				#pragma unroll
				for (int n = 0; n < TN; n++) {
					int b_shared_inc_col = tx * TN + n;
					sub_c[m][n] += a_shared[a_shared_inc_row][j] * b_shared[j][b_shared_inc_col];
				}
			}
		}
		__syncthreads();
	}
	#pragma unroll
	for (int m = 0; m < TM; m++) {
		int c_global_row = by * BM + ty * TM + m;
		if (c_global_row >= row) { break; }
		#pragma unroll
		for (int n = 0; n < TN; n += 4) {
			int c_global_col = bx * BN + tx * TN + n;
			if (c_global_col >= col) { break; }
			int c_global_site = c_global_row + c_global_col * row;
			for (int i = 0; i < 4; i++) {
				if (c_global_col + i >= col) { break; }
				c[c_global_site + i * row] = sub_c[m][n + i];
			}
		}
	}
}

void matmul_shared(MatrixXd& a, MatrixXd& b, MatrixXd& c) {
	double* aptr;
	double* bptr;
	double* cptr;
	size_t a_size = sizeof(double) * a.size();
	size_t b_size = sizeof(double) * b.size();
	size_t c_size = sizeof(double) * c.size();
	CHECK_CUDA_ERROR(hipMalloc(&aptr, a_size));
	CHECK_CUDA_ERROR(hipMalloc(&bptr, b_size));
	CHECK_CUDA_ERROR(hipMalloc(&cptr, c_size));
	CHECK_CUDA_ERROR(hipMemcpy(aptr, a.data(), a_size, hipMemcpyHostToDevice));
	CHECK_CUDA_ERROR(hipMemcpy(bptr, b.data(), b_size, hipMemcpyHostToDevice));
	dim3 blocksdim(16, 16, 1);
	dim3 gridsdim((b.cols() - 1) / 128 + 1, (a.rows() - 1) / 128 + 1, 1);
	multi_matrix_shared << <gridsdim, blocksdim >> > (aptr, bptr, cptr, a.rows(), b.cols(), a.cols());
	CHECK_CUDA_ERROR(hipDeviceSynchronize());
	CHECK_CUDA_ERROR(hipMemcpy(c.data(), cptr, c_size, hipMemcpyDeviceToHost));
	CHECK_CUDA_ERROR(hipFree(aptr));
	CHECK_CUDA_ERROR(hipFree(bptr));
	CHECK_CUDA_ERROR(hipFree(cptr));
	return ;
}

__global__ void add_matrix(double* a, double* b, double* c, int row, int col) {
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	if (x < col && y < row) {
		c[y + x * row] = a[y + x * row] + b[y + x * row];
	}
}

void matadd(MatrixXd& a, MatrixXd& b, MatrixXd& c) {
	double* aptr;
	double* bptr;
	double* cptr;
	size_t a_size = sizeof(double) * a.size();
	CHECK_CUDA_ERROR(hipMalloc(&aptr, a_size));
	CHECK_CUDA_ERROR(hipMalloc(&bptr, a_size));
	CHECK_CUDA_ERROR(hipMalloc(&cptr, a_size));
	CHECK_CUDA_ERROR(hipMemcpy(aptr, a.data(), a_size, hipMemcpyHostToDevice));
	CHECK_CUDA_ERROR(hipMemcpy(bptr, b.data(), a_size, hipMemcpyHostToDevice));
	dim3 blockdim(32, 32, 1);
	dim3 griddim((a.cols() - 1) / 32 + 1, (a.rows() - 1) / 32 + 1, 1);
	add_matrix << <griddim, blockdim >> > (aptr, bptr, cptr, a.rows(), a.cols());
	CHECK_CUDA_ERROR(hipDeviceSynchronize());
	CHECK_CUDA_ERROR(hipMemcpy(c.data(), cptr, a_size, hipMemcpyDeviceToHost));
	CHECK_CUDA_ERROR(hipFree(aptr));
	CHECK_CUDA_ERROR(hipFree(bptr));
	CHECK_CUDA_ERROR(hipFree(cptr));
	return ;
}