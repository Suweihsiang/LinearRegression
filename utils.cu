#include "hip/hip_runtime.h"
#include"utils.cuh"
#include<stdio.h>
#include<stdlib.h>

template void matmul<MatrixXd>(MatrixXd& a, MatrixXd& b, MatrixXd& c);
template void matmul<VectorXd>(MatrixXd& a, VectorXd& b, VectorXd& c);
template void matmul_shared<MatrixXd>(MatrixXd& a, MatrixXd& b, MatrixXd& c);
template void matmul_shared<VectorXd>(MatrixXd& a, VectorXd& b, VectorXd& c);

#define CHECK_CUDA_ERROR(val) Check_cuda_Error((val),__FILE__,__LINE__)
void Check_cuda_Error(hipError_t error, const char* const file, const int line) {
	if (error != hipSuccess) {
		printf("CUDA Error Occurs at %s, line %d : #%d(%s)\n", file, line, error, hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}
}

__global__ void multi_matrix(double* as, double* bs, double* cs, int row, int col, int k) {
	int r = threadIdx.y + blockDim.y * blockIdx.y;
	int c = threadIdx.x + blockDim.x * blockIdx.x;
	if (r < row && c < col) {
		double val = 0.0;
		for (int i = 0; i < k; i++) {
			val += as[r + i * row] * bs[c * k + i];
		}
		cs[r + c * row] = val;
	}
}

template<typename T>
void matmul(MatrixXd& a, T& b, T& c) {
	//create pointer
	double* as;
	double* bs;
	double* cs;
	//size of matrix
	size_t a_sz = sizeof(double) * a.size();
	size_t b_sz = sizeof(double) * b.size();
	size_t c_sz = sizeof(double) * c.size();
	//allocate memory to pointer
	CHECK_CUDA_ERROR(hipMalloc(&as, a_sz));
	CHECK_CUDA_ERROR(hipMalloc(&bs, b_sz));
	CHECK_CUDA_ERROR(hipMalloc(&cs, c_sz));
	//copy matrix data to pointer
	CHECK_CUDA_ERROR(hipMemcpy(as, a.data(), a_sz, hipMemcpyHostToDevice));
	CHECK_CUDA_ERROR(hipMemcpy(bs, b.data(), b_sz, hipMemcpyHostToDevice));
	//threads and block size
	dim3 threads_per_block(32, 32, 1);
	dim3 blocks_per_grid((b.cols() + 32 - 1) / 32, (a.rows() + 32 - 1) / 32, 1);
	multi_matrix << <blocks_per_grid, threads_per_block >> > (as, bs, cs, a.rows(), b.cols(), a.cols());
	CHECK_CUDA_ERROR(hipDeviceSynchronize());
	//copy pointer data to matrix
	CHECK_CUDA_ERROR(hipMemcpy(c.data(), cs, c_sz, hipMemcpyDeviceToHost));
	//release allocated memory
	CHECK_CUDA_ERROR(hipFree(as));
	CHECK_CUDA_ERROR(hipFree(bs));
	CHECK_CUDA_ERROR(hipFree(cs));
	return;
}

__global__ void multi_matrix_shared(double* a,double* b, double* c, int row, int col, int k) {
	const int BM = 128;
	const int BN = 128;
	const int BK = 8;
	const int TM = 8;
	const int TN = 8;

	__shared__ double a_shared[BM][BK];//a_shared is a 128*8 matrix
	__shared__ double b_shared[BK][BN];//b_shared is a 8*128 matrix
	double sub_c[TM][TN] = { 0.0 };//sub_c is a 8*8 matrix that save part of matrix multiply by a_shared and b_shared

	const int by = blockIdx.y;
	const int bx = blockIdx.x;
	const int ty = threadIdx.y;
	const int tx = threadIdx.x;
	const int tid = ty * blockDim.x + tx;//thread index
	//one thread construct part of a_shared that size is 1*4
	int a_shared_row = tid >> 1;
	int a_shared_col = (tid % 2) << 2;
	//one thread construct part of b_shared that size is 1*4
	int b_shared_row = tid >> 5;
	int b_shared_col = (tid % 32) << 2;

	int a_global_row = by * BM + a_shared_row;//the row that corresponding to pointer a's row
	int b_global_col = bx * BN + b_shared_col;//the column that corresponding to pointer b's column

	for (int bk = 0; bk < (k + BK - 1) / BK; bk++) {
		int a_global_col = bk * BK + a_shared_col;//the column that corresponding to pointer a's column
		int a_global_site = a_global_row + a_global_col * row;//the site that corresponding to pointer a's site

		int b_global_row = bk * BK + b_shared_row;//the row that corresponding to pointer b's row
		int b_global_site = b_global_row + b_global_col * k;//the site that corresponding to pointer b's site
		for (int i = 0; i < 4; i++) {//construct a_shared and b_shared matrix
			a_shared[a_shared_row][a_shared_col + i] = (a_global_row < row && a_global_col < k) ? a[a_global_site + i * row] : 0;
			b_shared[b_shared_row][b_shared_col + i] = (b_global_row < k && b_global_col < col) ? b[b_global_site + i * k] : 0;
		}
		__syncthreads();//every threads synchronous construct these two shared matrics

		#pragma unroll
		for (int j = 0; j < BK; j++) {
			#pragma unroll
			for (int m = 0; m < TM; m++) {
				int a_shared_inc_row = ty * TM + m;//the row in a_shared
				#pragma unroll
				for (int n = 0; n < TN; n++) {//calculate sub_c matrix
					int b_shared_inc_col = tx * TN + n;//the column in b_shared
					sub_c[m][n] += a_shared[a_shared_inc_row][j] * b_shared[j][b_shared_inc_col];//multiply two matrics
				}
			}
		}
		__syncthreads();//every threads synchronous multiply part of these two shared matrics to get 8*8 sub_c matrix
	}
	#pragma unroll
	for (int m = 0; m < TM; m++) {
		int c_global_row = by * BM + ty * TM + m;//the row that corresponding to pointer c's row
		if (c_global_row >= row) { break; }
		#pragma unroll
		for (int n = 0; n < TN; n += 4) {
			int c_global_col = bx * BN + tx * TN + n;//the column that corresponding to pointer c's column
			if (c_global_col >= col) { break; }
			int c_global_site = c_global_row + c_global_col * row; //the site that corresponding to pointer c's site
			for (int i = 0; i < 4; i++) {
				if (c_global_col + i >= col) { break; }
				c[c_global_site + i * row] = sub_c[m][n + i];//save the result of sub_c to pointer c
			}
		}
	}
}

template<typename T>
void matmul_shared(MatrixXd& a, T& b, T& c) {
	//create pointer
	double* aptr;
	double* bptr;
	double* cptr;
	//size of matrix
	size_t a_size = sizeof(double) * a.size();
	size_t b_size = sizeof(double) * b.size();
	size_t c_size = sizeof(double) * c.size();
	//allocate memory to pointer
	CHECK_CUDA_ERROR(hipMalloc(&aptr, a_size));
	CHECK_CUDA_ERROR(hipMalloc(&bptr, b_size));
	CHECK_CUDA_ERROR(hipMalloc(&cptr, c_size));
	//copy matrix data to pointer
	CHECK_CUDA_ERROR(hipMemcpy(aptr, a.data(), a_size, hipMemcpyHostToDevice));
	CHECK_CUDA_ERROR(hipMemcpy(bptr, b.data(), b_size, hipMemcpyHostToDevice));
	//thread and block size
	dim3 blocksdim(16, 16, 1);
	dim3 gridsdim((b.cols() - 1) / 128 + 1, (a.rows() - 1) / 128 + 1, 1);
	multi_matrix_shared << <gridsdim, blocksdim >> > (aptr, bptr, cptr, a.rows(), b.cols(), a.cols());
	CHECK_CUDA_ERROR(hipDeviceSynchronize());
	//copy pointer data to matrix
	CHECK_CUDA_ERROR(hipMemcpy(c.data(), cptr, c_size, hipMemcpyDeviceToHost));
	//release allocate memory
	CHECK_CUDA_ERROR(hipFree(aptr));
	CHECK_CUDA_ERROR(hipFree(bptr));
	CHECK_CUDA_ERROR(hipFree(cptr));
	return;
}


__global__ void multi_matvec_shared(double* a, double* b, double* c, int row, int k) {

	const int BK = 512;
	const int TM = 128;

	__shared__ double b_shared[BK];
	double sub_c[TM] = { 0.0 };

	const int bx = blockIdx.x;
	const int by = blockIdx.y;
	const int tx = threadIdx.x;
	const int ty = threadIdx.y;

	int a_global_row = 8 * by * blockDim.y + 8 * ty;

	for (int bk = 0; bk < (k - 1) / BK + 1; bk++) {
		for (int i = 0; i < 8; i++) {
			b_shared[8 * tx + i] = (bk * BK + 8 * tx + i < k) ? b[bk * BK + 8 * tx + i] : 0;
		}
		__syncthreads();
		#pragma unroll
		for (int j = 0; j < BK; j++) {
			if (j + bk * BK >= k) { break; }
			#pragma unroll
			for (int i = 0; i < 8; i++) {
				if (a_global_row + i >= row) { break; }
				sub_c[8 * ty + i] += a[a_global_row + i + row * (j + bk * BK)] * b_shared[j];
			}
		}
		__syncthreads();
	}
	int c_global_row = 8 * by * blockDim.y + 8 * ty;
	#pragma unroll
	for (int i = 0; i < 8; i++) {
		if (c_global_row + i >= row) { break; }
		c[c_global_row + i] = sub_c[8 * ty + i];
	}
}

void matvecmul_shared(MatrixXd& a, VectorXd& b, VectorXd& c) {
	double* aptr;
	double* bptr;
	double* cptr;
	size_t a_size = sizeof(double) * a.size();
	size_t b_size = sizeof(double) * b.size();
	size_t c_size = sizeof(double) * c.size();
	CHECK_CUDA_ERROR(hipMalloc(&aptr, a_size));
	CHECK_CUDA_ERROR(hipMalloc(&bptr, b_size));
	CHECK_CUDA_ERROR(hipMalloc(&cptr, c_size));
	CHECK_CUDA_ERROR(hipMemcpy(aptr, a.data(), a_size, hipMemcpyHostToDevice));
	CHECK_CUDA_ERROR(hipMemcpy(bptr, b.data(), b_size, hipMemcpyHostToDevice));
	dim3 blocksdim(64, 16, 1);
	dim3 gridsdim((a.cols() - 1) / 512 + 1, (a.rows() - 1) / 128 + 1, 1);
	multi_matvec_shared << <gridsdim, blocksdim >> > (aptr, bptr, cptr, a.rows(), a.cols());
	CHECK_CUDA_ERROR(hipDeviceSynchronize());
	CHECK_CUDA_ERROR(hipMemcpy(c.data(), cptr, c_size, hipMemcpyDeviceToHost));
	CHECK_CUDA_ERROR(hipFree(aptr));
	CHECK_CUDA_ERROR(hipFree(bptr));
	CHECK_CUDA_ERROR(hipFree(cptr));
	return;
}

__global__ void add_matrix(double* a, double* b, double* c, int row, int col) {
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	if (x < col && y < row) {
		c[y + x * row] = a[y + x * row] + b[y + x * row];
	}
}

void matadd(MatrixXd& a, MatrixXd& b, MatrixXd& c) {
	double* aptr;
	double* bptr;
	double* cptr;
	size_t a_size = sizeof(double) * a.size();
	CHECK_CUDA_ERROR(hipMalloc(&aptr, a_size));
	CHECK_CUDA_ERROR(hipMalloc(&bptr, a_size));
	CHECK_CUDA_ERROR(hipMalloc(&cptr, a_size));
	CHECK_CUDA_ERROR(hipMemcpy(aptr, a.data(), a_size, hipMemcpyHostToDevice));
	CHECK_CUDA_ERROR(hipMemcpy(bptr, b.data(), a_size, hipMemcpyHostToDevice));
	dim3 blockdim(32, 32, 1);
	dim3 griddim((a.cols() - 1) / 32 + 1, (a.rows() - 1) / 32 + 1, 1);
	add_matrix << <griddim, blockdim >> > (aptr, bptr, cptr, a.rows(), a.cols());
	CHECK_CUDA_ERROR(hipDeviceSynchronize());
	CHECK_CUDA_ERROR(hipMemcpy(c.data(), cptr, a_size, hipMemcpyDeviceToHost));
	CHECK_CUDA_ERROR(hipFree(aptr));
	CHECK_CUDA_ERROR(hipFree(bptr));
	CHECK_CUDA_ERROR(hipFree(cptr));
	return;
}